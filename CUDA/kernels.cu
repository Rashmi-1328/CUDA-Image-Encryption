//CUDA kernels and CUDA kernel-related function definitions


#include <hip/hip_runtime.h>
#include <cstdint>
#include <cstdio>
using namespace std;

// Warm-up Kernel
__global__ void KWarmUp()
{
    return;
}

// ENC::Permutation by Rotation
__global__ void ENC_RotatePerm(const uint8_t* __restrict__ in, uint8_t* __restrict__ out, const int* __restrict__ colRotate, const int* __restrict__ rowRotate)
{
    int colShift = colRotate[blockIdx.y];
    int rowShift = rowRotate[(blockIdx.x + colShift) % gridDim.x];
    int InDex = ((gridDim.y) * blockIdx.x + blockIdx.y) * blockDim.x + threadIdx.x;
    int OutDex = ((gridDim.y) * ((blockIdx.x + colShift) % gridDim.x) + (blockIdx.y + rowShift) % gridDim.y) * blockDim.x + threadIdx.x;
    out[OutDex] = in[InDex];
}

// DEC::Permutation by Rotation
__global__ void DEC_RotatePerm(const uint8_t* __restrict__ in, uint8_t* __restrict__ out, const int* __restrict__ colRotate, const int* __restrict__ rowRotate)
{
    int colShift = colRotate[blockIdx.y];
    int rowShift = rowRotate[(blockIdx.x + colShift) % gridDim.x];
    int InDex = ((gridDim.y) * blockIdx.x + blockIdx.y) * blockDim.x + threadIdx.x;
    int OutDex = ((gridDim.y) * ((blockIdx.x + colShift) % gridDim.x) + (blockIdx.y + rowShift) % gridDim.y) * blockDim.x + threadIdx.x;
    out[InDex] = in[OutDex];
}

// Diffusion (top-down)
__global__ void DIFF_TD(const uint8_t* __restrict__ in, uint8_t* __restrict__ out, const double* __restrict__ xRow, const double* __restrict__ yRow, const int rows, const double r)
{
    // Initialize parameters
    double x = xRow[blockIdx.x];
    double y = yRow[blockIdx.x];
    const int stride = gridDim.x * blockDim.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Each thread diffuses one channel of a column
    for (int i = 0; i < rows; i++, idx += stride)
    {
        x = r * (3 * y + 1) * x * (1 - x);
        y = r * (3 * x + 1) * y * (1 - y);
        out[idx] = in[idx] ^ (uint8_t)(x * 256);
    }
}

// ENC::SELF-XOR (left-right)
__global__ void ENC_XOR_LR(uint8_t* __restrict__ in, const int cols)
{
    // Initialize parameters
    int prev = cols * blockIdx.x * blockDim.x + threadIdx.x;
    int curr = prev + blockDim.x;

    // Each thread diffuses one channel of a row
    for (int i = 1; i < cols; i++)
    {
        in[curr] ^= in[prev];
        prev = curr;
        curr += blockDim.x;
    }
}

// DEC::SELF-XOR (left-right)
__global__ void DEC_XOR_LR(uint8_t* __restrict__ img, const int cols)
{
    // Initialize parameters
    int curr = cols * blockIdx.x * blockDim.x + threadIdx.x + (cols - 1) * blockDim.x;
    int next = curr - blockDim.x;

    // Each thread diffuses one channel of a row
    for (int i = 1; i < cols; i++)
    {
        img[curr] ^= img[next];
        curr = next;
        next -= blockDim.x;
    }
}

//Compute sum of image gray level values

__global__ void imageSum(uint8_t* __restrict__ img, uint32_t *sum)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x; 
  atomicAdd(sum, img[index]);
}

// Wrappers for kernel calls
extern "C" void kernel_WarmUp()
{
    KWarmUp <<<1, 1>>> ();
}

extern "C" void Wrap_RotatePerm(uint8_t * in, uint8_t * out, int* colRotate, int* rowRotate, const dim3 & grid, const dim3 & block, const int mode)
{
    if (mode == 1)
    {
        float time;
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
        
        ENC_RotatePerm <<<grid, block>>> (in, out, colRotate, rowRotate);
        ENC_RotatePerm <<<grid, block>>> (out, in, colRotate, rowRotate);
        
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
    
        std::printf("\nTime to permute:  %3.6f ms \n", time);
    }
    
    else
    {
        float time;
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
           
        DEC_RotatePerm <<<grid, block>>> (in, out, colRotate, rowRotate);
        DEC_RotatePerm <<<grid, block>>> (out, in, colRotate, rowRotate);
        
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
    
        std::printf("\nTime to unpermute:  %3.6f ms \n", time);
    }
}

extern "C" void Wrap_Diffusion(uint8_t * &in, uint8_t * &out, const double*& randRowX, const double*& randRowY, const int dim[], const double r, const int mode)
{
    // Set grid and block size
    const dim3 gridCol(dim[0], 1, 1);
    const dim3 gridRow(dim[1],1, 1);
    const dim3 block(dim[2], 1, 1);
    
    if (mode == 1)
    {
        float time;
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
        
        DIFF_TD <<<gridRow, block>>> (in, out, randRowX, randRowY, dim[0], r);
        ENC_XOR_LR <<<gridRow, block>>> (out, dim[0]);
        
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
    
        std::printf("\nTime to diffuse:  %3.6f ms \n", time);
    }

    else
    {
        float time;
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
        
        DEC_XOR_LR <<<gridRow, block>>> (in, dim[0]);
        DIFF_TD <<<gridRow, block>>> (in, out, randRowX, randRowY, dim[0], r);
        
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
    
        std::printf("\nTime to undiffuse:  %3.6f ms \n", time);
    }
}

extern "C" void Wrap_imageSum(uint8_t *&image_vec, uint32_t *sum, const int dim[])
{
  // Set grid and block size
  const dim3 grid((dim[0] * dim[1]), 1, 1);
  const dim3 block(dim[2], 1, 1);
  
  float time;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  
  imageSum<<<grid, block>>>(image_vec, sum); 
    
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
    
  std::printf("\nTime to calculate sum:  %3.6f ms \n", time);
}


