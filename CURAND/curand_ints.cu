#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <assert.h>

__global__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
} 

__global__ void generate( hiprandState* globalState, int * result, int *max, int *min, int count ) 
{
    int ind = threadIdx.x;
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform( &localState );
    globalState[ind] = localState; 

    if (ind < count)

        result[ind] = truncf(*min +(*max - *min)*RANDOM);
}

int main( int argc, char** argv) 
{
    int N = 32; // no of random numbers to be generated

    int MIN = 10; // max range of random number
    int MAX = 100; // min range of random number

    dim3 tpb(N,1,1);
    hiprandState* devStates;
    hipMalloc ( &devStates, N*sizeof( hiprandState ) );

    // setup seeds
    setup_kernel <<< 1, tpb >>> ( devStates, time(NULL) );

    int *d_result, *h_result;

    hipMalloc(&d_result, N * sizeof(int));
    h_result = (int *)malloc(N * sizeof(int));

    int *d_max, *h_max, *d_min, *h_min;

    hipMalloc(&d_max, sizeof(int));
    h_max = (int *)malloc(sizeof(int));

    hipMalloc(&d_min, sizeof(int));
    h_min = (int *)malloc(sizeof(int));

    *h_max =MAX;
    *h_min =MIN;

    hipMemcpy(d_max, h_max, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_min, h_min, sizeof(int), hipMemcpyHostToDevice);

    // generate random numbers
    generate <<< 1, tpb >>> ( devStates, d_result, d_max, d_min, N );

    hipMemcpy(h_result, d_result, N * sizeof(float), hipMemcpyDeviceToHost);

      for (int i = 0; i < N; i++)
    printf("random number= %d\n", h_result[i]);

    return 0;
}
