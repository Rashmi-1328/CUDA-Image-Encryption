#include <iostream> /*For IO*/
#include <cstdlib>  /*For malloc()*/
#include <hip/hip_runtime.h>   /*For CUDA*/

#include <hiprand/hiprand.h> /*For CURAND*/
#include <hiprand/hiprand_kernel.h>
#include <cstdio>   /*For printf()*/
#include <cstdint>  /*For standard variable support*/

using namespace std;

int main()
{
  int n=10;
  int i;
  hiprandGenerator_t gen;
  unsigned int *devData, *hostData;
  
  /*Allocate n integers on the host*/
  hostData=(unsigned int*)malloc(n * sizeof(unsigned int));
  
  /*Initializing hostData*/
  for(int i=0;i<n;++i)
  {
    hostData[i]=0;
  }
  
  /*Allocate n unsigned ints on device*/
  hipMalloc((void**) &devData , n * sizeof(unsigned int));
  
  /*Create an MTGP Host Generator*/
  hiprandCreateGenerator(&gen , HIPRAND_RNG_PSEUDO_MTGP32);
  
  /*Set seed*/
  hiprandSetPseudoRandomGeneratorSeed(gen , 1);

  /*Generate n ints on the device*/
  hiprandGenerate(gen, devData, n);
  
  /*Copy device memory to host*/
  hipMemcpy(hostData, devData, n * sizeof(unsigned int), hipMemcpyDeviceToHost);
  
  cout<<"\nRandom draws: \n";
  for(i = 0; i < n; ++i) 
  {
    printf(" %d",hostData[i]%1024);
  }
  cout<<"\n";
  
  /*Clean up*/
  hiprandDestroyGenerator(gen);
  
  return 0;
}

