  
#include <hip/hip_runtime.h>
#include <cstdint>
 
    __global__ void ArMapImg(uint8_t *in, uint8_t *out)
    {
        int nx = (2*blockIdx.x + blockIdx.y) % gridDim.x;
        int ny = (blockIdx.x + blockIdx.y) % gridDim.y;
        int InDex = ((gridDim.x)*blockIdx.y + blockIdx.x) * 3  + threadIdx.x;
        int OutDex = ((gridDim.x)*ny + nx) * 3 + threadIdx.x;
        out[OutDex] = in[InDex];
    }

    __global__ void WarmUp()
    {
      return;
    }
    
    __global__ void FracXor(uint8_t *in, uint8_t *out, uint8_t *fractal)
    {
        int idx = blockIdx.x * 3 + threadIdx.x;
        out[idx] = in[idx]^fractal[idx];
    } 

    __global__ void Enc_GenCatMap(uint8_t *in, uint8_t *out, uint16_t *colRotate, uint16_t *rowRotate)
    {
        int colShift = colRotate[blockIdx.y];
        int rowShift = rowRotate[(blockIdx.x + colShift)%gridDim.x];
        int InDex    = ((gridDim.y)*blockIdx.x + blockIdx.y) * 3  + threadIdx.x;
        int OutDex   = ((gridDim.y)*((blockIdx.x + colShift)%gridDim.x) + (blockIdx.y + rowShift)%gridDim.y) * 3  + threadIdx.x;
        out[OutDex]  = in[InDex];
    }

    __global__ void Dec_GenCatMap(uint8_t *in, uint8_t *out, uint16_t *colRotate, uint16_t *rowRotate)
    {
        int colShift = colRotate[blockIdx.y];
        int rowShift = rowRotate[(blockIdx.x + colShift)%gridDim.x];
        int OutDex   = ((gridDim.y)*blockIdx.x + blockIdx.y) * 3  + threadIdx.x;
        int InDex    = ((gridDim.y)*((blockIdx.x + colShift)%gridDim.x) + (blockIdx.y + rowShift)%gridDim.y) * 3  + threadIdx.x;
        out[OutDex]  = in[InDex];
    }
   
   

   extern "C" void run_ArMapImg(uint8_t *in, uint8_t *out,dim3 blocks,dim3 block_size)
   {
     ArMapImg<<<blocks,block_size>>>(in,out);
     hipDeviceSynchronize();
   }

   extern "C" void run_WarmUp(dim3 blocks,dim3 block_size)
   {
     WarmUp<<<blocks,block_size>>>();
     hipDeviceSynchronize();
   }
  
  extern "C" void run_FracXor(uint8_t *in,uint8_t *out,uint8_t *fractal,dim3 blocks,dim3 block_size)
  {
    FracXor<<<blocks,block_size>>>(in,out,fractal);
    hipDeviceSynchronize();  
  }

  extern "C" void run_EncGenCatMap(uint8_t *in,uint8_t *out,uint16_t *colRotate,uint16_t *rowRotate,dim3 blocks,dim3 block_size)
  {
    Enc_GenCatMap<<<blocks,block_size>>>(in,out,colRotate,rowRotate);
    hipDeviceSynchronize();
  }
  
  extern "C" void run_DecGenCatMap(uint8_t *in,uint8_t *out,uint16_t *colRotate,uint16_t *rowRotate,dim3 blocks,dim3 block_size)
  {
     Dec_GenCatMap<<<blocks,block_size>>>(in,out,colRotate,rowRotate);
     hipDeviceSynchronize();    
  }
