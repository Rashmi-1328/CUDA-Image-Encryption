  
#include <hip/hip_runtime.h>
#include <iostream>
  #include <cstdio>
  #include <cstdint>
  using namespace std;
   
  /**
   * CUDA kernel. Gets GPU ready to perform computation. Helps achieve accurate GPU benchmarking. Takes the number of blocks and block size as arguments
   */  
    
    __global__ void WarmUp()
    {
      return;
    }
    
    
  /**
   * CUDA kernel. Rotates image rows and columns. Based on Arnold Cat Map. Accepts images of dimensions N x N and N x M. Takes two N X M 1D vectors and the number of blocks and block size as arguments
   */
    
    __global__ void Enc_GenCatMap(uint8_t* in, uint8_t* out, const uint32_t* __restrict__ colRotate, const uint32_t* __restrict__ rowRotate)
    {
        int colShift = colRotate[blockIdx.y];
        int rowShift = rowRotate[(blockIdx.x + colShift)%gridDim.x];
        int InDex    = ((gridDim.y)*blockIdx.x + blockIdx.y) * blockDim.x  + threadIdx.x;
        int OutDex   = ((gridDim.y)*((blockIdx.x + colShift)%gridDim.x) + (blockIdx.y + rowShift)%gridDim.y) * blockDim.x  + threadIdx.x;
        out[OutDex]  = in[InDex];
    }

  /**
   *  CUDA kernel. Unrotates image rows and columns. Based on Arnold Cat Map. Accepts images of dimensions N x N and N x M. Takes two N X M 1D vectors and the number of blocks and block size as arguments
   */ 
   __global__ void Dec_GenCatMap(uint8_t* in, uint8_t* out, const uint32_t* __restrict__ colRotate, const uint32_t* __restrict__ rowRotate)
    {
        int colShift = colRotate[blockIdx.y];
        int rowShift = rowRotate[(blockIdx.x + colShift)%gridDim.x];
        int OutDex   = ((gridDim.y)*blockIdx.x + blockIdx.y) * blockDim.x  + threadIdx.x;
        int InDex    = ((gridDim.y)*((blockIdx.x + colShift)%gridDim.x) + (blockIdx.y + rowShift)%gridDim.y) * blockDim.x  + threadIdx.x;
        out[OutDex]  = in[InDex];
    }
   
    
   /**
    * CUDA kernel. Swaps image rows and columns. Accepts images of dimensions N x N and N x M. Takes two N X M 1D vectors and the number of blocks and block size as arguments
    */
   __global__ void encRowColSwap(uint8_t* img_in,uint8_t* img_out, const uint32_t* __restrict__ rowSwapLUT, const uint32_t* __restrict__ colSwapLUT)
   {
      int blockId = blockIdx.y * gridDim.x + blockIdx.x;
      int threadId = blockId * blockDim.x + threadIdx.x;
      
      int gray_level_index_in = threadId;
      int row = rowSwapLUT[blockIdx.x];
      int col = colSwapLUT[blockIdx.y];
      int pixel_index_out = row * gridDim.y + col;
      int gray_level_index_out = pixel_index_out * blockDim.x + threadIdx.x;
      img_out[gray_level_index_in] = img_in[gray_level_index_out];
      
   }
   
  /**
   * CUDA kernel. Unswaps image rows and columns. Accepts images of dimensions N x N and N x M. Takes two N X M 1D vectors and the number of blocks and block size as arguments
   */
  __global__ void decRowColSwap(uint8_t* img_in,uint8_t* img_out, const uint32_t* __restrict__ rowSwapLUT,const uint32_t* __restrict__ colSwapLUT)
  {
    int blockId= blockIdx.y * gridDim.x + blockIdx.x;
    int threadId = blockId * blockDim.x + threadIdx.x;
    
    int gray_level_index_in = threadId;
    int row = rowSwapLUT[blockIdx.x];
    int col = colSwapLUT[blockIdx.y];
    int pixel_index_out = row * gridDim.y + col;
    int gray_level_index_out = pixel_index_out * blockDim.x + threadIdx.x;
    img_out[gray_level_index_out] = img_in[gray_level_index_in];
  }   

  /**
   * CUDA kernel wrapper function. Gets GPU ready to perform computation. Helps achieve accurate GPU benchmarking. Takes the number of blocks and block size as arguments
   */
  extern "C" void run_WarmUp(dim3 blocks,dim3 block_size)
  {
     
    WarmUp<<<blocks,block_size>>>();
     
  }
  
  /**
   * CUDA kernel wrapper function. Rotates image rows and columns. Based on Arnold Cat Map. Accepts images of dimensions N x N and N x M. Takes two N X M 1D vectors and the number of blocks and block size as arguments
   */
  extern "C" void run_EncGenCatMap(uint8_t* in,uint8_t* out,const uint32_t* __restrict__ colRotate, const uint32_t* __restrict__ rowRotate,dim3 blocks,dim3 block_size)
  { 
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    Enc_GenCatMap<<<blocks,block_size>>>(in,out,colRotate,rowRotate);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    printf("\nTime to rotate:  %3.6f ms \n", time);
    
  }
  
  /**
   *  CUDA kernel wrapper function. Unrotates image rows and columns. Based on Arnold Cat Map. Accepts images of dimensions N x N and N x M. Takes two N X M 1D vectors and the number of blocks and block size as arguments
   */
  extern "C" void run_DecGenCatMap(uint8_t* in,uint8_t* out,const uint32_t* __restrict__ colRotate, const uint32_t* __restrict__ rowRotate,dim3 blocks,dim3 block_size)
  {
     float time;
     hipEvent_t start, stop;
     hipEventCreate(&start);
     hipEventCreate(&stop);
     hipEventRecord(start, 0);
     
     Dec_GenCatMap<<<blocks,block_size>>>(in,out,colRotate,rowRotate);
     
     hipEventRecord(stop, 0);
     hipEventSynchronize(stop);
     hipEventElapsedTime(&time, start, stop);
     
     printf("\nTime to unrotate:  %3.6f ms \n", time);
        
  }

  /**
   * CUDA kernel wrapper function. Swaps image rows and columns. Accepts images of dimensions N x N and N x M. Takes two N X M 1D vectors and the number of blocks and block size as arguments
   */
  extern "C" void run_encRowColSwap(uint8_t* img_in,uint8_t* img_out, const uint32_t* __restrict__ rowSwapLUT, const uint32_t* __restrict__ colSwapLUT,dim3 blocks,dim3 block_size)
  {
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    encRowColSwap<<<blocks, block_size>>>(img_in,img_out,rowSwapLUT,colSwapLUT);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    
    printf("\nTime to swap:  %3.6f ms \n", time);
    
  }
  
  /**
   * CUDA kernel wrapper function. nswaps image rows and columns. Accepts images of dimensions N x N and N x M. Takes two N X M 1D vectors and the number of blocks and block size as arguments
   */
  extern "C" void run_decRowColSwap(uint8_t* img_in,uint8_t* img_out,const uint32_t* __restrict__ rowSwapLUT,const uint32_t* __restrict__ colSwapLUT,dim3 blocks,dim3 block_size)
  {
     float time;
     hipEvent_t start, stop;
     hipEventCreate(&start);
     hipEventCreate(&stop);
     hipEventRecord(start, 0);
    
     decRowColSwap<<<blocks,block_size>>>(img_in,img_out,rowSwapLUT,colSwapLUT);
     
     hipEventRecord(stop, 0);
     hipEventSynchronize(stop);
     hipEventElapsedTime(&time, start, stop);
     
     printf("\nTime to unswap:  %3.6f ms \n", time);
  }

