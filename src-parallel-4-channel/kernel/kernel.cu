  
#include <hip/hip_runtime.h>
#include <iostream>
  #include <cstdio>
  #include <cstdint>
  using namespace std;
   
      
    
    __global__ void WarmUp()
    {
      return;
    }
    
    
    __global__ void Enc_GenCatMap(uint8_t* in, uint8_t* out, const uint32_t* __restrict__ colRotate, const uint32_t* __restrict__ rowRotate)
    {
        int colShift = colRotate[blockIdx.y];
        int rowShift = rowRotate[(blockIdx.x + colShift)%gridDim.x];
        int InDex    = ((gridDim.y)*blockIdx.x + blockIdx.y) * blockDim.x  + threadIdx.x;
        int OutDex   = ((gridDim.y)*((blockIdx.x + colShift)%gridDim.x) + (blockIdx.y + rowShift)%gridDim.y) * blockDim.x  + threadIdx.x;
        out[OutDex]  = in[InDex];
    }

    __global__ void Dec_GenCatMap(uint8_t* in, uint8_t* out, const uint32_t* __restrict__ colRotate, const uint32_t* __restrict__ rowRotate)
    {
        int colShift = colRotate[blockIdx.y];
        int rowShift = rowRotate[(blockIdx.x + colShift)%gridDim.x];
        int OutDex   = ((gridDim.y)*blockIdx.x + blockIdx.y) * blockDim.x  + threadIdx.x;
        int InDex    = ((gridDim.y)*((blockIdx.x + colShift)%gridDim.x) + (blockIdx.y + rowShift)%gridDim.y) * blockDim.x  + threadIdx.x;
        out[OutDex]  = in[InDex];
    }
   
    
  
   __global__ void encRowColSwap(uint8_t* img_in,uint8_t* img_out, const uint32_t* __restrict__ rowSwapLUT, const uint32_t* __restrict__ colSwapLUT)
   {
      int blockId = blockIdx.y * gridDim.x + blockIdx.x;
      int threadId = blockId * blockDim.x + threadIdx.x;
      
      int gray_level_index_in = threadId;
      int row = rowSwapLUT[blockIdx.x];
      int col = colSwapLUT[blockIdx.y];
      int pixel_index_out = row * gridDim.y + col;
      int gray_level_index_out = pixel_index_out * blockDim.x + threadIdx.x;
      img_out[gray_level_index_in] = img_in[gray_level_index_out];
      
   }
   
  __global__ void decRowColSwap(uint8_t* img_in,uint8_t* img_out, const uint32_t* __restrict__ rowSwapLUT,const uint32_t* __restrict__ colSwapLUT)
  {
    int blockId= blockIdx.y * gridDim.x + blockIdx.x;
    int threadId = blockId * blockDim.x + threadIdx.x;
    
    int gray_level_index_in = threadId;
    int row = rowSwapLUT[blockIdx.x];
    int col = colSwapLUT[blockIdx.y];
    int pixel_index_out = row * gridDim.y + col;
    int gray_level_index_out = pixel_index_out * blockDim.x + threadIdx.x;
    img_out[gray_level_index_out] = img_in[gray_level_index_in];
  }   

  /*Gets GPU ready to perform computation. Helps achieve accurate GPU benchmarking*/
  extern "C" void run_WarmUp(dim3 blocks,dim3 block_size)
  {
     
    WarmUp<<<blocks,block_size>>>();
     
  }
  
  /*Rotates image rows and columns. Based on Arnold Cat Map. Accepts images of dimensions N x N and N x M*/
  extern "C" void run_EncGenCatMap(uint8_t* in,uint8_t* out,const uint32_t* __restrict__ colRotate, const uint32_t* __restrict__ rowRotate,dim3 blocks,dim3 block_size)
  { 
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    Enc_GenCatMap<<<blocks,block_size>>>(in,out,colRotate,rowRotate);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    printf("\nTime to rotate:  %3.6f ms \n", time);
    
  }
  
  /*Unrotates image rows and columns. Based on Arnold Cat Map. Accepts images of dimensions N x N and N x M*/
  extern "C" void run_DecGenCatMap(uint8_t* in,uint8_t* out,const uint32_t* __restrict__ colRotate, const uint32_t* __restrict__ rowRotate,dim3 blocks,dim3 block_size)
  {
     float time;
     hipEvent_t start, stop;
     hipEventCreate(&start);
     hipEventCreate(&stop);
     hipEventRecord(start, 0);
     
     Dec_GenCatMap<<<blocks,block_size>>>(in,out,colRotate,rowRotate);
     
     hipEventRecord(stop, 0);
     hipEventSynchronize(stop);
     hipEventElapsedTime(&time, start, stop);
     
     printf("\nTime to unrotate:  %3.6f ms \n", time);
        
  }

  /*Swaps image rows and columns. Accepts images of dimensions N x N and N x M*/
  extern "C" void run_encRowColSwap(uint8_t* img_in,uint8_t* img_out, const uint32_t* __restrict__ rowSwapLUT, const uint32_t* __restrict__ colSwapLUT,dim3 blocks,dim3 block_size)
  {
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    encRowColSwap<<<blocks, block_size>>>(img_in,img_out,rowSwapLUT,colSwapLUT);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    
    printf("\nTime to swap:  %3.6f ms \n", time);
    
  }
  
  /*Unswaps image rows and columns. Accepts images of dimensions N x N and N x M*/
  extern "C" void run_decRowColSwap(uint8_t* img_in,uint8_t* img_out,const uint32_t* __restrict__ rowSwapLUT,const uint32_t* __restrict__ colSwapLUT,dim3 blocks,dim3 block_size)
  {
     float time;
     hipEvent_t start, stop;
     hipEventCreate(&start);
     hipEventCreate(&stop);
     hipEventRecord(start, 0);
    
     decRowColSwap<<<blocks,block_size>>>(img_in,img_out,rowSwapLUT,colSwapLUT);
     
     hipEventRecord(stop, 0);
     hipEventSynchronize(stop);
     hipEventElapsedTime(&time, start, stop);
     
     printf("\nTime to unswap:  %3.6f ms \n", time);
  }
